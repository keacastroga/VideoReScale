#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <sys/time.h>

#include <hip/hip_runtime_api.h>

using namespace std;
using namespace cv;



__global__ void CUDAScale(uchar *pixels, uchar *newPixels, int newRows, int newCols, int rowLen, int newRowLen);

int main(int argc, char **argv)
{
    int rows, cols, channels, rowLen, newRows, newCols, newRowLen;
    hipError_t err = hipSuccess;
    if (argc != 4)
    {
        printf("usage: VideoReScale.out <Video_Path> <scale factor> <Output_path>\n");
        return -1;
    }

    VideoCapture cap(argv[1]);
    if (!cap.isOpened())
    {
        cout << "Error opening video stream or file" << endl;
        return -1;
    }

    VideoWriter out;
    double factor = stod(argv[2]);
    double fps = cap.get(CAP_PROP_FPS);
    //int fourcc = static_cast<int>(cap.get(CAP_PROP_FOURCC));
    int fourcc = VideoWriter::fourcc('H', '2', '6', '4'); 
    rows = (int)cap.get(CAP_PROP_FRAME_HEIGHT);
    cols = (int)cap.get(CAP_PROP_FRAME_WIDTH);
    channels = 3;
    rowLen = channels * cols;
    newRows = rows * factor;
    newCols = cols * factor;
    newRowLen = channels * newCols;

    Size S = Size(newCols, newRows);

    size_t sizeOrig = sizeof(uchar) * cols * channels * rows;
    size_t sizeNew = sizeof(uchar) * newCols * channels * newRows;
    unsigned char *h_newPixels;
    h_newPixels = (uchar *)malloc(sizeNew);
    
    
    float *d_newPixels = NULL;
    err = hipMalloc((void **)&d_newPixels, sizeNew);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device new pixels array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *d_pixels = NULL;
    err = hipMalloc((void **)&d_pixels, sizeOrig);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device original pixels array (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int threadsPerBlock = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * 2;
    int blocksPerGrid =(newCols + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    out.open(argv[3], fourcc, fps, S);
    if (!out.isOpened())
    {
        cout << "Could not open the output video for write: " << endl;
        return -1;
    }

    Mat frame;
    Mat outFrame(newRows, newCols, CV_8UC3, h_newPixels);
    struct timeval tval_before, tval_after, tval_result;
    gettimeofday(&tval_before, NULL);
    while (1)
    {
        cap >> frame;
        if (frame.empty())
            break;
        hipMemcpy(d_pixels, frame.data, sizeOrig, hipMemcpyHostToDevice);
        CUDAScale<<<blocksPerGrid, threadsPerBlock>>>((uchar *)d_pixels, (uchar *)d_newPixels, newRows, newCols, rowLen, newRowLen);
        hipMemcpy(h_newPixels, d_newPixels, sizeNew, hipMemcpyDeviceToHost);
        out << outFrame;
    }
    gettimeofday(&tval_after, NULL);
    timersub(&tval_after, &tval_before, &tval_result);
    printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);
    hipFree(d_pixels);
    hipFree(d_newPixels);
    free(h_newPixels);
    out.release();
    cap.release();
    return 0;
}


__global__ void CUDAScale(uchar *pixels, uchar *newPixels, int newRows, int newCols, int rowLen, int newRowLen){
    int scaledCol = blockDim.x * blockIdx.x + threadIdx.x;
    int origCol;
    uchar *pixel1, *pixel2, *pixel3, *pixel4;
    uchar *newPixel;
    if (scaledCol < newCols)
    {
        origCol = scaledCol * 6;
        for (int scaledRow = 0; scaledRow < newRows; scaledRow++)
        {
            pixel1 = pixels + origCol + scaledRow * 2 * rowLen;
            pixel2 = pixel1 + 3;
            pixel3 = pixel1 + rowLen;
            pixel4 = pixel3 + 3;
            newPixel = newPixels + scaledCol * 3 + scaledRow * newRowLen;
            *(newPixel) = (*pixel1 + *pixel2 + *pixel3 + *pixel4) / 4;
            *(newPixel + 1) = (*(pixel1 + 1) + *(pixel2 + 1) + *(pixel3 + 1) + *(pixel4 + 1)) / 4;
            *(newPixel + 2) = (*(pixel1 + 2) + *(pixel2 + 2) + *(pixel3 + 2) + *(pixel4 + 2)) / 4;
        }
    }
}
